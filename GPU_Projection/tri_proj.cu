#include "mex.h"
#include "matrix.h"
#include "gpu/mxGPUArray.h"
#include "tet_proj.cuh"

static bool mexInitialized;
static mxArray * gpuCanary;
static hipStream_t stream;
static void uninit();
static char const * const errInputId = "qtimes:InvalidInput";
static char const * const errCudaId = "qtimes:CudaError";
static char const * const errCudaMsg = "qtimes encountered a CUDA error.";

//initialize stream
static bool init() {
    if (!mexInitialized || !mxGPUIsValidGPUData(gpuCanary)) {
        // Initialize the MATLAB GPU API if not already initialized.
        if (mxInitGPU() != MX_GPU_SUCCESS) {
            return false;
        }

        const size_t one = 1;
        hipStreamCreate(&stream); // Note that a pointer must be passed to `cudaCreateStream`.

        mxGPUArray * canary = mxGPUCreateGPUArray(1, &one, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
        gpuCanary = mxGPUCreateMxArrayOnGPU(canary);
        mxGPUDestroyGPUArray(canary);
        mexMakeArrayPersistent(gpuCanary);
        mexInitialized = true;
        mexAtExit(uninit);
    }
    return true;
}

//destroy stream
static void uninit() {
    if (mexInitialized) {
        mxDestroyArray(gpuCanary);
        hipStreamDestroy(stream); // Note that a value, not a pointer, is passed to `cudaDestroyStream`.
    }
}

void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{   //inputs:
    // points: dim x n_p
    // v0: dim x n_t
    // e_mat:  dim x 3xn_t (vertex i - vertex 0)
    // e_dot_mat: 3x3xn_t
    // e_dot_inv_mat: 3x3xn_t
    // is_degenerate: n_tx1
    // Initialize solver
    if (!init()) {
        mexErrMsgIdAndTxt(errCudaId, errCudaMsg);
    }

    if (nrhs < 2) {
        mexErrMsgIdAndTxt(errInputId, "Must provide an operation and input matrix.");
    }


	// Throw an error if the 1st input is not a GPU array.
    if (!mxIsGPUArray(prhs[0]) || !mxGPUIsValidGPUData(prhs[0])) {
        mexErrMsgIdAndTxt(errInputId, "Inputs to tet proj must be of type double gpuArray.");
    }

    // Unwrap input to an mxGPUArray (must be real double). Repeat for e_mat, E_dot_mat, e_dot_inv_mat
    const mxGPUArray *points = mxGPUCreateFromMxArray(prhs[0]);
    const mxGPUArray *v0 = mxGPUCreateFromMxArray(prhs[1]);
    const mxGPUArray *e_mat = mxGPUCreateFromMxArray(prhs[2]);
    const mxGPUArray *e_dot_mat = mxGPUCreateFromMxArray(prhs[3]);
    const mxGPUArray *e_dot_inv_mat = mxGPUCreateFromMxArray(prhs[4]);
    const mxGPUArray *is_degenerate =mxGPUCreateFromMxArray(prhs[5]);
    // do this for all of the inputs
    if (mxGPUGetClassID(v0) != mxDOUBLE_CLASS || mxGPUGetComplexity(v0) != mxREAL) {
        mexErrMsgIdAndTxt(errInputId, "Inputs to tet proj must be of type double gpuArray.");
    }
    //repeat for each input
    size_t nDimv0 = mxGPUGetNumberOfDimensions(v0);
    if (nDimv0 != 2) {
        mexErrMsgIdAndTxt(errInputId, "batchop operates on 2D or 3D arrays only.");
    }
    const size_t * dim_points = mxGPUGetDimensions(points);
    const size_t * dim_v0 = mxGPUGetDimensions(v0);
    const size_t num_tets = dim_v0[1];
    const size_t num_points = dim_points[1];
    const size_t dim = dim_v0[0];
    const size_t * dim_e_mat = mxGPUGetDimensions(e_mat);

    if(dim_e_mat[2] != num_tets){
        mexErrMsgIdAndTxt(errInputId, "3D matrix has incorrect dimensions.");
    }
    
    size_t num_threads = 128;
    dim3 num_blocks(num_points);
    size_t shared_size = num_threads * (sizeof(double) + sizeof(size_t));

    // create outputs. Repeat for all. Use int for the outputindices. mxINT32_CLASS
    size_t dimResultDists[] = {num_points, 1};
    size_t dimResultWeights[] = {num_points,3};
    mxGPUArray * result_dists = mxGPUCreateGPUArray(2,dimResultDists, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    mxGPUArray * result_idxs = mxGPUCreateGPUArray(2,dimResultDists, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    mxGPUArray * result_weights = mxGPUCreateGPUArray(2,dimResultWeights, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);

    // call function
    // TODO: FIGURE OUT A BETTER WAY TO INITIALIZE THE DIMENSION (NOT HARD CODE 6). add stream back in.
    tet_kernels::GeneralizedTetrahedronProjectionKernel<6, double>
                    <<<num_blocks, num_threads, shared_size, stream>>>(
                            static_cast<const double *>(mxGPUGetDataReadOnly(points)),
                            num_tets,
                            static_cast<const double *>(mxGPUGetDataReadOnly(v0)),
                            static_cast<const double *>(mxGPUGetDataReadOnly(e_mat)),
                            static_cast<const double *>(mxGPUGetDataReadOnly(e_dot_mat)),
                            static_cast<const double *>(mxGPUGetDataReadOnly(e_dot_inv_mat)),
                            static_cast<const double*>(mxGPUGetDataReadOnly(is_degenerate)),
                            static_cast<double *>(mxGPUGetData(result_dists)), 
                            static_cast<int *>(mxGPUGetData(result_idxs)),
                            static_cast<double *>(mxGPUGetData(result_weights)));
                        

    plhs[0] = mxGPUCreateMxArrayOnGPU(result_dists);
    plhs[1] = mxGPUCreateMxArrayOnGPU(result_idxs);
    plhs[2] = mxGPUCreateMxArrayOnGPU(result_weights);
    mxGPUDestroyGPUArray(result_dists);
    mxGPUDestroyGPUArray(result_idxs);
    mxGPUDestroyGPUArray(result_weights);

//     mxFree((void *)dim_points);
//     mxFree((void *)nDimv0);
//     mxFree((void *)dim_e_mat);
    mxGPUDestroyGPUArray(points);
    mxGPUDestroyGPUArray(v0);
    mxGPUDestroyGPUArray(e_mat);
    mxGPUDestroyGPUArray(e_dot_mat);
    mxGPUDestroyGPUArray(e_dot_inv_mat);
    mxGPUDestroyGPUArray(is_degenerate);

    
}